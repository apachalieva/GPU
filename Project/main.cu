#include "hip/hip_runtime.h"
extern "C"{
    #include "helper.h"
    #include "visual.h"
    #include "init.h"
    #include "uvp.h"
    #include "boundary_val.h"
    #include "sor.h"
}
#include <stdio.h>
#include <string.h>

#define PARAMF "cavity.dat"
#define VISUAF "visual/sim"

#define OBSTACLE 0
#define FLUID 1
#define INFLOW 2

/**
 * The main operation reads the configuration file, initializes the scenario and
 * contains the main loop. So here are the individual steps of the algorithm:
 *
 * - read the program configuration file using read_parameters()
 * - set up the matrices (arrays) needed using the matrix() command
 * - create the initial setup init_uvp(), init_flag(), output_uvp()
 * - perform the main loop
 * - trailer: destroy memory allocated and do some statistics
 *
 * The layout of the grid is decribed by the first figure below, the enumeration
 * of the whole grid is given by the second figure. All the unknowns corresond
 * to a two dimensional degree of freedom layout, so they are not stored in
 * arrays, but in a matrix.
 *
 * @image html grid.jpg
 *
 * @image html whole-grid.jpg
 *
 * Within the main loop the following big steps are done (for some of the 
 * operations a definition is defined already within uvp.h):
 *
 * - calculate_dt() Determine the maximal time step size.
 * - boundaryvalues() Set the boundary values for the next time step.
 * - calculate_fg() Determine the values of F and G (diffusion and confection).
 *   This is the right hand side of the pressure equation and used later on for
 *   the time step transition.
 * - calculate_rs()
 * - Iterate the pressure poisson equation until the residual becomes smaller
 *   than eps or the maximal number of iterations is performed. Within the
 *   iteration loop the operation sor() is used.
 * - calculate_uv() Calculate the velocity at the next time step.
 */

// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ###
// ###


#include "aux.h"
#include <iostream>
#include <math.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA


__device__ float cuda_diff_x(float a, float b, int x, int w)
{

	if (x+1<w)
	{
		return (a - b);
	}
	else
	{
		return 0.0f;
	}
	
}

__device__ float cuda_diff_y(float a, float b, int y, int h)
{

	if (y+1<h)
	{
		return (a - b);
	}
	else
	{
		return 0.0f;
	}
	
}

__global__ void global_grad(float *imgIn, float *v1, float *v2, int w, int h, int nc, int n)
{

	int ind = threadIdx.x + blockDim.x * blockIdx.x;

	int x, y, ch;	

	ch = (int)(ind) / (int)(w*h);
	y = (ind - ch*w*h) / (int)w;
	x = (ind - ch*w*h) % (int)w;

	if (ind<n)
	{ 

		v1[ind] = cuda_diff_x(imgIn[ind+1], imgIn[ind], x, w);
		v2[ind] = cuda_diff_y(imgIn[ind+w], imgIn[ind], y, h);

	}

	

}


__device__ float cuda_div_x(float a, float b, int x, int w)
{
		if ((x+1<w) && (x>0))
		{
			return (a - b);
		}
		else if (x+1<w)
		{
			return (a - 0);
		}
		else if (x>0)
		{
			return (0 - b);
		}
		else
		{
			return 0.;convert_layered_to_mat_int
		}
}


__device__ float cuda_div_y(float a, float b, int y, int h)
{
		if ((y+1<h) && (y>0))
		{
			return (a - b);
		}
		else if (y+1<h)
		{convert_layered_to_mat_int
			return (a - 0);
		}
		else if (y>0)
		{
			return (0 - b);
		}
		else
		{
			return 0.;
		}
}

__global__ void global_div(float *v1, float *v2, float *imgOut, int w, int h, int nc, int n)
{

	int ind = threadIdx.x + blockDim.x * blockIdx.x;

	int x, y, ch;

	ch = (int)(ind) / (int)(w*h);
	y = (ind - ch*w*h) / (int)w;
	x = (ind - ch*w*h) % (int)w;

	if ((ind<n) && (ind-w>=0) && (ind-1>convert_layered_to_mat_int=0)) 
	{ 	
		imgOut[ind] = cuda_div_x(v1[ind], v1[ind-1], x, w) + cuda_div_y(v2[ind], v2[ind-w], y, h);
	}

}




__global__ void global_norm(float *imgIn, float *imgOut, int w, int h, int n)
{
	int ind = threadIdx.x + blockDim.x * blockIdx.x;
	if (ind<n)
	{ 
		imgOut[ind] = imgIn[ind]*imgIn[ind];
		imgOut[ind] += imgIn[ind+w*h]*imgIn[ind+w*h];
		imgOut[ind] += imgIn[ind+2*w*h]*imgIn[ind+2*w*h];
		imgOut[ind] = sqrtf(imgOut[iconvert_layered_to_mat_intnd]);
	}
}

__device__ int check_color(float *c, float r, float g, float b)
{

	float  eps = 0.0001;


	if ( (fabsf(r-c[0])<eps) && (fabsf(g-c[1])<eps) &&  (fabsf(b-c[2])<eps) )
	{
		return 1;
	}
	else
	{
		return 0;
	}	
}

__global__ void global_detect_domain(float *imgIn, int *imgDomain, int w, int h, int n)
{

	float c[3] = {1.0f, 0.0f, 0.0f};
	
	// For looping around a pixel
	int neighbour[8]={1, -1, w, -w, -w-1, -w+1, w-1, w+1};

	int ind = threadIdx.x + blockDim.x * blockIdx.x;

	int x, y, ch;

	ch = (int)(ind) / (int)(w*h);
	y = (ind - ch*w*h) / (int)w;
	x = (ind - ch*w*h) % (int)w;

	if (ind<n)
	{

		if (check_color(c, imgIn[ind], imgIn[ind+w*h], imgIn[ind+2*w*h]))
		{
			imgDomain[ind] = FLUID;
			for (int i=0; i<8; i++)
			{
				//TODO: Check if ind+neighbour[i] is in the domain!
				if ( check_color(c, imgIn[ind+neighbour[i]], imgIn[ind+w*h+neighbour[i]], imgIn[ind+2*w*h+neighbour[i]]) != 1 )
				{
					imgDomain[ind+neighbour[i]] = INFLOW;
				}
			}
		}
		else
		{
			imgDomain[ind] = OBSTACLE;
		}
	}
}

//======================================================================================================================================================
//==============================================================CFD CODE================================================================================

int mainCFD(int argc, char** args, float *imgU, float *imgV, int *imgDomain){
	double Re, UI, VI, PI, GX, GY, t_end, xlength, ylength, dt, dx, dy, alpha, omg, tau, eps, dt_value, t, res,dp;
	double **U, **V, **P, **F, **G, **RS;
	int n, step, it, imax, jmax, itermax, pb;
	int fluid_cells;		/* Number of fluid cells in our geometry */
	char problem[10];		/* Problem name, file name */
	int boundaries[4];
	char *fname;

	int **Flag;			/* Flagflield matrix */

	if(argc>=2)
		fname=args[1];
	else
		fname = PARAMF;

	read_parameters(fname, &Re, &UI, &VI, &PI, &GX, &GY, &t_end, &xlength, &ylength, &dt, &dx, &dy, &imax, &jmax, &alpha, &omg, &tau, &itermax, &eps, &dt_value, boundaries, &dp, &pb);
	/* setting of the problem */
	switch (pb){
		case 0:	strcpy(problem,"karman");
		break;
		case 1:	strcpy(problem,"shear");
		break;
		case 2:	strcpy(problem,"step");
		break;
		default: strcpy(problem,"none");
		}


	fluid_cells = imax*jmax;

	/* Allocate Flag matrix */
	Flag = imatrix( 0, imax+1, 0, jmax+1 );

	/* should we change the dimension of the matrices in order to save space? */
	U = matrix ( 0 , imax+1 , 0 , jmax+1 );
	V = matrix ( 0 , imax+1 , 0 , jmax+1 );
	P = matrix ( 0 , imax+1 , 0 , jmax+1 );

	F = matrix ( 0 , imax , 0 , jmax );
	G = matrix ( 0 , imax , 0 , jmax );
	RS = matrix ( 0 , imax , 0 , jmax );

	init_flag( problem, imax, jmax, &fluid_cells, Flag );
	init_uvp(UI, VI, PI, imax, jmax, U, V, P, Flag, problem);

	t=.0;
	n=0;
	step=0;

	while( t <= t_end ){
		if( tau > 0 ) calculate_dt(Re, tau, &dt, dx, dy, imax, jmax, U, V);

		boundaryvalues( imax, jmax, U, V, boundaries, Flag );
		/* special inflow boundaries */
		spec_boundary_val( problem, imax, jmax, U, V, Re, dp, ylength);

		/* calculate new values for F and G */
		calculate_fg( Re, GX, GY, alpha, dt, dx, dy, imax, jmax, U, V, F, G, Flag );
		/* calculate right hand side */
		calculate_rs( dt, dx, dy, imax, jmax, F, G, RS, Flag );

		it = 0;
		res = 10000.0;
		while( it < itermax && fabs(res) > eps ){
			sor( omg, dx, dy, imax, jmax, fluid_cells, P, RS, Flag, &res, problem, dp );
			it++;
		}

		printf("[%d: %f] dt: %f, sor iterations: %d \n", n, t, dt, it);

		if( it == itermax )
		    printf( "    WARNING: Maximum number of iterations reached.\n" );

		calculate_uv( dt, dx, dy, imax, jmax, U, V, F, G, P, Flag );

		t += dt;
		n++;

		if(step*dt_value <= t){
			/* output vtk file for visualization */
			write_vtkFile( VISUAF, n, xlength, ylength, imax, jmax, dx, dy, U, V, P );
			step++;
		}

	}

	printf("Problem: %s\n", problem );
	printf( "imax = %d, jmax = %d\n", imax, jmax );
	printf( "Number of fluid cells = %d\n", fluid_cells );
	printf( "Reynolds number: %f\n", Re);


	/* free memory */
	free_matrix(U,0,imax+1,0,jmax+1);
	free_matrix(V,0,imax+1,0,jmax+1);
	free_matrix(P,0,imax+1,0,jmax+1);

	free_matrix(F,0,imax,0,jmax);
	free_matrix(G,0,imax,0,jmax);
	free_matrix(RS,0,imax,0,jmax);

	free_imatrix( Flag, 0, imax+1, 0, jmax+1 );

	return 0;
}

//======================================================================================================================================================

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> <gamma> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    




    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    //cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
	float *imgOut = new float[(size_t)w*h*nc];
	float *v1 = new float[(size_t)w*h*nc];
	float *v2 = new float[(size_t)w*h*nc];
	float *imgVorticity = new float[(size_t)w*h*mOut.channels()];
	int *imgDomain = new int[(size_t)w*h];
	// TODO: Temporarly we consider just a grayscale inpainting
	float *imgU = new float[(size_t)w*h];
	float *imgV = new float[(size_t)w*h];


    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;convert_layered_to_mat_int
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);


    Timer timer; timer.start();
    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###

	int n = w*h*nc, n2=w*h;

	// Calculate gradient

	// allocate GPU memory
	float *gpu_In, *gpu_v1, *gpu_v2, *gpu_Out, *gpu_Vorticity, *gpu_U, *gpu_V;
	int *gpu_Domain;

	hipMalloc(&gpu_In, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_v1, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_v2, n*sizeof(float));
	CUDA_CHECK;
	// TODO: Temporarly we consider just a grayscale inpainting
	hipMalloc(&gpu_U, w*h*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_V, w*h*sizeof(float));
	CUDA_CHECK;


	// copy host memory to device
	hipMemcpy(gpu_In, imgIn, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_v1, v1, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_v2, v2, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;

	// launch kernel
	dim3 block = dim3(128,1,1);
	
	dim3 grid = dim3((n + block.x - 1) / block.x, 1, 1);
	global_grad <<<grid,block>>> (gpu_In, gpu_v1, gpu_v2, w, h, nc, n);
	global_norm <<<grid,block>>> (gpu_v1, gpu_V, w, h, w*h);
	global_norm <<<grid,block>>> (gpu_v2, gpu_U, w, h, w*h);

	// copy result back to host (CPU) memory
	hipMemcpy(v1, gpu_v1, n * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;
	hipMemcpy(v2, gpu_v2, n * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;
	hipMemcpy(imgU, gpu_U, w*h * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;
	hipMemcpy(imgV, gpu_V, w*h * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;

	// free device (GPU) memory
	hipFree(gpu_In);
	CUDA_CHECK;
	hipFree(gpu_v1);
	CUDA_CHECK;
	hipFree(gpu_v2);
	CUDA_CHECK;
	hipFree(gpu_U);
	CUDA_CHECK;
	hipFree(gpu_V);
	CUDA_CHECK;

	// Invert the V values according t: V = -dI/dx
	// TODO: Temporarly we consider just a grayscale inpainting 
	for (int i=0; i<w*h; i++)
	{
		imgV[i] = -imgV[i];
	}

	
	// Calculate divergence of a gradient

	hipMalloc(&gpu_v1, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_v2, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_Out, n*sizeof(float));
	CUDA_CHECK;

	// copy host memory to device
	hipMemcpy(gpu_v1, v1, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_v2, v2, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_Out, imgOut, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;

	// launch kernel
	global_div <<<grid,block>>> (gpu_v1, gpu_v2, gpu_Out, w, h, nc, n);

	// copy result back to host (CPU) memory
	hipMemcpy(imgOut, gpu_Out, n * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;

	// free device (GPU) memory
	hipFree(gpu_v1);
	CUDA_CHECK;
	hipFree(gpu_v2);
	CUDA_CHECK;
	hipFree(gpu_Out);
	CUDA_CHECK;


	// Calculate norm	
	// allocate GPU memory

	hipMalloc(&gpu_In, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_Vorticity, n2*sizeof(float));
	CUDA_CHECK;

	// copy host memory to device
	hipMemcpy(gpu_In, imgOut, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_Vorticity, imgVorticity, n2*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;

	// launch kernel
	global_norm <<<grid,block>>> (gpu_In, gpu_Vorticity, w, h, n2);

	// copy result back to host (CPU) memory
	hipMemcpy(imgVorticity, gpu_Vorticity, n2 * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK;

	// free device (GPU) memory
	hipFree(gpu_In);
	CUDA_CHECK;
	hipFree(gpu_Vorticity);
	CUDA_CHECK;



	// Calculate the inpainting domain	
	// allocate GPU memory

	hipMalloc(&gpu_In, n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&gpu_Domain, w*h*sizeof(int));
	CUDA_CHECK;

	// copy host memory to device
	hipMemcpy(gpu_In, imgIn, n*sizeof(float), hipMemcpyHostToDevice);
	CUDA_CHECK;
	hipMemcpy(gpu_Domain, imgDomain, w*h*sizeof(int), hipMemcpyHostToDevice);
	CUDA_CHECK;

	// launch kernel
	global_detect_domain <<<grid,block>>> (gpu_In, gpu_Domain, w, h, w*h);

	// copy result back to host (CPU) memory
	hipMemcpy(imgDomain, gpu_Domain, w*h * sizeof(int), hipMemcpyDeviceToHost );
	CUDA_CHECK;

	// free device (GPU) memory
	hipFree(gpu_In);
	CUDA_CHECK;
	hipFree(gpu_Domain);
	CUDA_CHECK;



    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;


	mainCFD(argc, argv, imgU, imgV, imgDomain);


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat_int(mOut, imgDomain);
    showImage("Output1", mOut, 100+w+40, 100);


    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgVorticity;
    delete[] imgDomain;
    delete[] v1;
    delete[] v2;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



