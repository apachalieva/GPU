#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2013/2014, March 3 - April 4
// ###
// ###
// ### Evgeny Strekalovskiy, Maria Klodt, Jan Stuehmer, Mohamed Souiai
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <cstdlib>
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
using std::stringstream;
using std::cerr;
using std::cout;
using std::endl;
using std::string;




// parameter processing
template<>
bool getParam<bool>(std::string param, bool &var, int argc, char **argv)
{
    const char *c_param = param.c_str();
    for(int i=argc-1; i>=1; i--)
    {
        if (argv[i][0]!='-') continue;
        if (strcmp(argv[i]+1, c_param)==0)
        {
            if (!(i+1<argc) || argv[i+1][0]=='-') { var = true; return true; }
            std::stringstream ss;
            ss << argv[i+1];
            ss >> var;
            return (bool)ss;
        }
    }
    return false;
}




// opencv helpers
void convert_layered_to_interleaved(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[(nc-1-c) + nc*(x + (size_t)w*y)] = aIn[x + (size_t)w*y + nOmega*c];
            }
        }
    }
}

void convert_interleaved_to_layered(float *aOut, const float *aIn, int w, int h, int nc)
{
    if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[x + (size_t)w*y + nOmega*c] = aIn[(nc-1-c) + nc*(x + (size_t)w*y)];
            }
        }
    }
}

void convert_mat_to_layered(float *aOut, const cv::Mat &mIn)
{
    convert_interleaved_to_layered(aOut, (float*)mIn.data, mIn.cols, mIn.rows, mIn.channels());
}

void convert_layered_to_mat(cv::Mat &mOut, const float *aIn)
{
    convert_layered_to_interleaved((float*)mOut.data, aIn, mOut.cols, mOut.rows, mOut.channels());
}

void normalize_for_display(float *aOut, const float *aIn, int n)
{
    float mn = FLT_MAX;
    float mx = -FLT_MAX;
    for(int i=0; i<n; i++)
    {
        float val = aIn[i];
        if (val<mn) mn = val;
        if (val>mx) mx = val;
    }
    if (n==0) { mn = 0; mx = 1; }
    for(int i=0; i<n; i++)
    {
        float val = aIn[i];
        val = (val-mn)/(mx-mn);
        aOut[i] = val;
    }
}

void showImage(string title, const cv::Mat &mat, int x, int y)
{
    const char *wTitle = title.c_str();
    cv::namedWindow(wTitle, CV_WINDOW_AUTOSIZE);
    cvMoveWindow(wTitle, x, y);
    cv::imshow(wTitle, mat);
}

float noise(float sigma)
{
    float x1 = (float)rand()/RAND_MAX;
    float x2 = (float)rand()/RAND_MAX;
    return sigma * sqrtf(-2*log(std::max(x1,0.000001f)))*cosf(2*M_PI*x2);
}
void addNoise(cv::Mat &m, float sigma)
{
    float *data = (float*)m.data;
    int w = m.cols;
    int h = m.rows;
    int nc = m.channels();
    size_t n = (size_t)w*h*nc;
    for(size_t i=0; i<n; i++)
    {
        data[i] += noise(sigma);
    }
}



// cuda error checking
string prev_file = "";
size_t prev_line = 0;
void cuda_check(string file, size_t line)
{
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(hipError_t) << " (" << hipError_t << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        exit(1);
    }
    prev_file = file;
    prev_line = line;
}
